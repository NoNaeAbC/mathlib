#include "hip/hip_runtime.h"
//
// Created by af on 25.01.21.
//
#include "amathlib.h"
#include <iostream>

__global__ void kernel(double height, double width) {
	MatrixDouble4X4 m;
	//m.identity();
	//MatrixDouble4X4 m2({height, width, height, width}, {height, width, height, width},
	//				   {height, width, height, width}, { height, width, height, width });
	//MatrixDouble4X4 m3 = m2 * m;
}


int main() {

#if defined(__global__)
	std::cout << "cuda" << std::endl;

#endif
#if defined(__device__)
	std::cout << "cuda" << std::endl;

#endif

	// Run kernel
	dim3 blockDim(1, 1, 1);
	dim3 gridDim((1 + blockDim.x - 1) / blockDim.x, (1 + blockDim.y - 1) / blockDim.y, 1);
	kernel<<< gridDim, blockDim, 0 >>>(1, 1);

}
