#include "hip/hip_runtime.h"
//
// Created by af on 25.01.21.
//
#include <iostream>

#define AML_CUDA

#include "amathlib.h"

__global__ void kernel(double width, double height, int accuracy, int *results) {
	int i = blockIdx.x * blockDim.x + threadIdx.x; //width
	if (i >= (int) width) {
		return;
	}
	int j = blockIdx.y * blockDim.y + threadIdx.y; //height
	if (j >= (int) height) {
		return;
	}
	results[i + j * (int) width] = accuracy;
	results[i + j * (int) width] = 0;
	if (!(i >= width || j >= height)) {

		CU_Complex64 c = {CU_AML::mapLinear((double) j, 0.0, (double) height, -1.5, 0.5),
						  CU_AML::mapLinear((double) i, 0.0, (double) width, -1.0, 1.0)};
		CU_Complex64 z = c;
		for (int x = 0; x < accuracy; ++x) {
			z = z * z + c;
			if (z.abs_gt(2)) {
				results[i + j * (int) width] = accuracy;
				break;
			}
		}
	}
}


int main() {
	const int width = 250;
	const int height = 80;
	const int accuracy = 100000;

	int *deviceResults;
	hipMalloc(&deviceResults, width * height * sizeof(int));

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
				   (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernel<<< numBlocks, threadsPerBlock>>>(width, height, accuracy, deviceResults);
	std::cout << " : " << width << " : " << std::endl;

	int *results = (int *) malloc(width * height * sizeof(int));

	hipMemcpy(results, deviceResults, width * height * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(deviceResults);
	for (int x = 0; x < height; x++) {
		for (int y = 0; y < width; y++) {
			if (results[y + x * (int) width] >= accuracy) {
				std::cout << " ";
			} else if (results[y + x * (int) width] == 10) UNLIKELY {
				std::cout << ".";
			} else LIKELY {
				std::cout << "#";
			}
		}
		std::cout << "\n";
	}

	free(results);

}
